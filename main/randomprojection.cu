#include "hip/hip_runtime.h"
// Online C++ compiler to run C++ program online
#include <iostream>
#include <vector>
#include <cstdlib>   // for rand()
#include <ctime>     // for time()
#include <random>
#include <cmath>
#include <cfloat>
#include <numeric>
#include <algorithm>
#include <iomanip>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <cstdint>
#include <chrono>
#include "absl/status/status.h"
#include "absl/status/statusor.h"
#include "absl/log/log.h"
#include "absl/strings/str_cat.h"
#include "absl/types/span.h"
#include "util.hpp"


absl::Status CudaStatus(hipError_t code) {
  if (code != hipSuccess) {
    const char *error = hipGetErrorString(code);
    return absl::InvalidArgumentError(absl::StrCat("Cuda error: ", error));
  }
  return absl::OkStatus();
}
#define RETURN_IF_ERROR(expr) do {absl::Status _status = (expr);if (!_status.ok()) return _status;} while (0)

#define RET_CUDA(x) RETURN_IF_ERROR(CudaStatus(x))

#define RET_CUBLAS(stat) do { \
    if ((stat) != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        return absl::InternalError("cuBLAS call failed."); \
    } \
} while (0)

absl::Status CheckHasGPU(bool print_info) {
  static absl::Status status = [&]() -> absl::Status {
    int driver_version = 0;
    RET_CUDA(hipDriverGetVersion(&driver_version));
    if (driver_version == 0) {
      return absl::InvalidArgumentError("No matching cuda driver found");
    }
    hipDeviceProp_t prop;
    RET_CUDA(hipGetDeviceProperties(&prop, 0));
    if (print_info) {
      LOG(INFO) << "Using CUDA device: " << prop.name
                << " (driver:" << driver_version << ")";
    }
    return absl::OkStatus();
  }();
  return status;
}

//warm-up kernel
__global__ void warmup() {}

__global__ void RandomColumnGenerationKernel(int* total_col_indices,
                                             int* shuffle_buffer, 
                                             int num_cols, 
                                             int selected_features_count, 
                                             int num_proj, 
                                             unsigned long long seed) {

    int proj_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (proj_id >= num_proj) return;

    // Initialize RNG
    hiprandState state;
    hiprand_init(seed, proj_id, 0, &state);

    // Each thread gets its own slice of shared memory
    int* local_indices = shuffle_buffer + proj_id * num_cols;

    // Fill array 0..(num_cols - 1)
    for (int i = 0; i < num_cols; ++i) {
        local_indices[i] = i;
    }

    __syncthreads(); // sync before shuffle if threads cooperate

    // Shuffle the last selected_features_count elements
    for (int i = num_cols - 1; i >= num_cols - selected_features_count; --i) {
        int j = hiprand(&state) % (i + 1);
        int tmp = local_indices[i];
        local_indices[i] = local_indices[j];
        local_indices[j] = tmp;
    }
    __syncthreads(); // sync before writing results if threads cooperate
    // Write results to global memory
    int offset = proj_id * selected_features_count;
    for (int i = 0; i < selected_features_count; ++i) {
        total_col_indices[offset + i] = local_indices[num_cols - selected_features_count + i];
    }
}


__global__ void ColumnAddProjectionKernel(
  // [num_total_rows * num_cols]
  // [num_rows * num_cols]
  // [num_cols * num_proj]
  // [num_rows * num_proj]
    const float* __restrict__ dataset,            
    const int* __restrict__ flat_col_data,      
    float* projected,               
    int num_rows,
    int num_cols,
    int num_proj, 
    int selected_features_count) 

{
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // projection index
    int row_start = blockIdx.y * blockDim.y + threadIdx.y;

    // Stride over rows to avoid grid.y overflow
    for (int row = row_start; row < num_rows; row += gridDim.y * blockDim.y) {
        if (col < num_proj) {
            float sum = 0.0f;
            for (int i = 0; i < selected_features_count; ++i) {
                int feature_idx = flat_col_data[col * selected_features_count + i];
                if (feature_idx < num_cols) {
                    sum += dataset[feature_idx * num_rows + row];  // column-major access
                }
            }
            projected[col * num_rows + row] = sum;  // column-major output
        }
    }
}



void ApplyProjectionColumnADD (const float* flat_data,
                               float** d_col_add_projected_out,
                               float* GPU_Col_Add_values,
                               int* total_col_indices,
                               const int num_rows,
                               const int num_cols,
                               const int num_proj,
                               const int selected_features_count,
                               double& elapsed_ms,
                               bool verbose
                              ){

// Warm-up launch, The first kernel won’t be artificially inflated by setup costs
  warmup<<<1, 1>>>();
  hipDeviceSynchronize();

  int total_dataset_size = num_rows * num_cols;
  int total_col_dataset_size = selected_features_count * num_proj;
  int result_size = num_rows * num_proj;
  std::cout << std::endl;

///////////////////////////////////////////////////Debug/////////////////////////////////////
  if (verbose) {
    std::cout << "Passed col add data from main to GPU function: " << std::endl;
    std::cout << "rows: " << num_rows << std::endl;
    std::cout << "cols: " << num_cols << std::endl;
    std::cout << "proj: " << num_proj << std::endl;
  }
  ///////////////////////////////////////////////////Debug/////////////////////////////////////

  //Allocate device memory
  float *d_flat_data = nullptr;
  float *d_col_add_projected = nullptr;
  int *d_flat_col_data = nullptr;
  int* d_shuffle_buffer = nullptr;

  hipMalloc((void **)&d_flat_data, total_dataset_size * sizeof(float));
  hipMalloc((void **)&d_flat_col_data, total_col_dataset_size * sizeof(int));                              
  hipMalloc((void **)&d_col_add_projected, result_size * sizeof(float));
  hipMalloc((void **)&d_shuffle_buffer, num_proj * num_cols * sizeof(int));

  //Copy dataset to device
  hipMemcpy(d_flat_data, flat_data, total_dataset_size * sizeof(float), hipMemcpyHostToDevice);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      std::cerr << "CUDA ERROR: " << hipGetErrorString(err) << std::endl;
  }

  int threads_per_block = 256;
  int num_blocks = (num_proj + threads_per_block - 1) / threads_per_block;

  unsigned long long seed = static_cast<unsigned long long>(time(NULL));
  RandomColumnGenerationKernel<<<num_blocks, threads_per_block>>>(d_flat_col_data,
                                                d_shuffle_buffer,
                                                num_cols, 
                                                selected_features_count,
                                                num_proj, 
                                                seed);
  hipDeviceSynchronize();  // Ensure kernel finishes

  //Copy the generated column data from device to host
  hipMemcpy(total_col_indices, d_flat_col_data, num_proj * selected_features_count * sizeof(int), hipMemcpyDeviceToHost);
  //hipMemcpy(d_flat_col_data, total_col_indices, num_proj * selected_features_count * sizeof(int), hipMemcpyHostToDevice);  

    // Launch CUDA kernel
  int block_size_x = 16;
  int block_size_y = 16;
  // //int threads_per_block = block_size_x * block_size_y;
  dim3 block_size(block_size_x, block_size_y); 
  dim3 grid_size((num_proj + block_size_x - 1) / block_size_x,
          min((num_rows + block_size_y - 1) / block_size_y, 32768)); 

  auto startA = std::chrono::high_resolution_clock::now();
  ColumnAddProjectionKernel<<<grid_size, block_size>>>(d_flat_data,
                                                       d_flat_col_data,   
                                                       d_col_add_projected,
                                                       num_rows, 
                                                       num_cols,
                                                       num_proj,
                                                       selected_features_count);
  hipDeviceSynchronize();  // Ensure kernel finishes
  auto endA = std::chrono::high_resolution_clock::now();
  elapsed_ms = std::chrono::duration<double, std::milli>(endA - startA).count();

  *d_col_add_projected_out = d_col_add_projected;  // <-- pass it back

  hipMemcpy(GPU_Col_Add_values, d_col_add_projected, num_rows * num_proj * sizeof(float), hipMemcpyDeviceToHost);

//For debugging

  hipPeekAtLastError();
  // Free device memory
  hipFree(d_flat_data);
  hipFree(d_flat_col_data);
  //hipFree(d_col_add_projected); Don't free here if this is not the end.
  hipFree(d_shuffle_buffer);
}
